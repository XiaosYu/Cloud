#include "hip/hip_runtime.h"  
    
__global__ void addKernel(int* c, const int* a, const int* b);


__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
