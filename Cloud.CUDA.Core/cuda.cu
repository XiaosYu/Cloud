#include "hip/hip_runtime.h"
// cuda.cpp : ���� DLL Ӧ�ó���ĵ���������
//

#include "pch.h"
#include "hip/hip_runtime.h"  
#include "" 
__global__ void addKernel(int* c, const int* a, const int* b);

//�������  
int _stdcall ArrayAdd(int c[], int a[], int b[], int size)
{
	int result = -1;
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// ѡ���������е�GPU  
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		result = 1;
		goto Error;
	}

	// ��GPU��Ϊ����dev_a��dev_b��dev_c�����ڴ�ռ�.  
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 2;
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 3;
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		result = 4;
		goto Error;
	}

	// �������ڴ渴�����ݵ�GPU�ڴ���.  
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		result = 5;
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		result = 6;
		goto Error;
	}

	// ����GPU�ں˺���  
	addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

	// ����hipDeviceSynchronize�ȴ�GPU�ں˺���ִ����ɲ��ҷ����������κδ�����Ϣ  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		result = 7;
		goto Error;
	}

	// ��GPU�ڴ��и������ݵ������ڴ���  
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		result = 8;
		goto Error;
	}

	result = 0;

	// ����CUDA�豸�����˳�֮ǰ�������hipDeviceReset  
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 9;
	}

Error:
	//�ͷ��豸�б�����ռ�ڴ�  
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return result;

}
